#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    int N = 8;
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // Initialize a and b
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // Copy a and b to the device
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Run the kernel
    add<<<1, N>>>(dev_a, dev_b, dev_c);

    // Copy c back to the host
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free the memory on the device
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
